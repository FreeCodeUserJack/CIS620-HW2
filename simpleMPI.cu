#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */


/* Simple example demonstrating how to use MPI with CUDA
*
*  Generate some random numbers on one node.
*  Dispatch them to all nodes.
*  Compute their square root on each node's GPU.
*  Compute the average of the results using MPI.
*
*  simpleMPI.cu: GPU part, compiled with nvcc
*/

#include <iostream>
using std::cerr;
using std::endl;

#include "simpleMPI.h"

// Error handling macro
#define CUDA_CHECK(call) \
    if((call) != hipSuccess) { \
        hipError_t err = hipGetLastError(); \
        cerr << "CUDA error calling \""#call"\", code is " << err << endl; \
        my_abort(err); }


// Device code
// Very simple GPU Kernel that computes square roots of input numbers
__global__ void simpleMPIKernel(float *inputA, float *inputB, float *output)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    output[tid] = sqrt(inputA[tid] * inputA[tid] + inputB[tid] * inputB[tid]);
}


// Initialize an array with random data (between 0 and 1)
void initData(float *data, int dataSize)
{
    for (int i = 0; i < dataSize; i++)
    {
        data[i] = (float)rand() / RAND_MAX;
    }
}

// CUDA computation on each node
// No MPI here, only CUDA
void computeGPU(float *hostDataA, float *hostDataB, int blockSize, int gridSize)
{
    int dataSize = blockSize * gridSize;

    // Allocate data on GPU memory
    float *deviceInputDataA = NULL;
	float *deviceInputDataB = NULL;
    CUDA_CHECK(hipMalloc((void **)&deviceInputDataA, dataSize * sizeof(float)));
	CUDA_CHECK(hipMalloc((void **)&deviceInputDataB, dataSize * sizeof(float)));

    float *deviceOutputData = NULL;
    CUDA_CHECK(hipMalloc((void **)&deviceOutputData, dataSize * sizeof(float)));

    // Copy to GPU memory
    CUDA_CHECK(hipMemcpy(deviceInputDataA, hostDataA, dataSize * sizeof(float), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(deviceInputDataB, hostDataB, dataSize * sizeof(float), hipMemcpyHostToDevice));

    // Run kernel
    simpleMPIKernel<<<gridSize, blockSize>>>(deviceInputDataA, deviceInputDataB, deviceOutputData);

    // Copy data back to CPU memory //hostDataA
    CUDA_CHECK(hipMemcpy(hostDataA, deviceOutputData, dataSize *sizeof(float), hipMemcpyDeviceToHost));

    // Free GPU memory
    CUDA_CHECK(hipFree(deviceInputDataA));
	CUDA_CHECK(hipFree(deviceInputDataB));
    CUDA_CHECK(hipFree(deviceOutputData));
}

float maxr(float *data, int size)
{
    float max = data[0];

    for (int i = 1; i < size; i++)
    {
        if (data[i] > max)
			max = data[i];
    }

    return max;
}
